#include "hip/hip_runtime.h"
// Copyright (C) 2011  Monash University
// Copyright (C) 2011  Dmitri Nikulin
//
// Permission is hereby granted, free of charge, to any person
// obtaining a copy of this software and associated documentation
// files (the "Software"), to deal in the Software without
// restriction, including without limitation the rights to use,
// copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the
// Software is furnished to do so, subject to the following
// conditions:
//
// The above copyright notice and this permission notice shall be
// included in all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
// EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES
// OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
// NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT
// HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY,
// WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR
// OTHER DEALINGS IN THE SOFTWARE.

// Undo namespace damage done by CUDA's nvcc.
#undef isfinite
#undef isnan

#include <cvd/image_io.h>

#include <hip/hip_runtime.h>

#include <fstream>
#include <iomanip>
#include <iostream>
#include <vector>

// Use time() because nvcc isn't compatible with boost.
#include <ctime>

#define X_OFF            8
#define Y_OFF            7
#define FAST_RING        9
#define FAST_THRESH     40
#define REPEAT       10000

// Maximum number of corners.
#define FAST_COUNT (1 << 18)

// Number of threads per 1D group.
#define NTHREADS 512

// Declare 1-byte read-only texture object.
texture<uchar1, 2, hipReadModeElementType> static testImage;

__device__ int mask_test(uint x16) {
    // Duplicate bit pattern to simulate barrel shift.
    uint const x = (x16 | (x16 << 16));

    // Accumulator.
    uint x1 = x;

    // AND against down-shifts.
    #pragma unroll
    for (uint i = 1; i < FAST_RING; i++)
        x1 &= (x >> i);

    // Return of 1 here proves that FAST_RING
    // consecutive bits were 1.
    return (x1 != 0);
}

__global__ void fast1_kernel(
    int2 * corners,
    int  * icorner
) {

    // Calculate (x,y) of center pixel.
    int const x = ((blockIdx.x * blockDim.x) + threadIdx.x + X_OFF);
    int const y = ((blockIdx.y * blockDim.y) + threadIdx.y + Y_OFF);

    // Read center pixel, upcast to int.
    int const p00 = tex2D(testImage, x, y).x;

    // Include generated code here.
    // Checks ring of pixels, and populates the boolean "isCorner".
    #include "fast1_gen.cu"

    if (isCorner) {
        // Atomically append to corner buffer.
        int const icorn = atomicAdd(icorner, 1);
        if ((icorn >= 0) && (icorn < FAST_COUNT))
            corners[icorn] = make_int2(x, y);
    }
}

__global__ void fast2_kernel(
    int2 const * i_corners,
    int  const * i_ncorners,
    int2       * o_corners,
    int        * o_ncorners
) {

    // Find input offset.
    int const idx = ((blockIdx.x * blockDim.x) + threadIdx.x);
    if ((idx < 0) || (idx >= i_ncorners[0]))
        return;

    // Read (x,y) of center pixel.
    int2 const xy = i_corners[idx];
    int const x = xy.x;
    int const y = xy.y;

    // Read center pixel, upcast to int.
    int const p00 = tex2D(testImage, x, y).x;

    // Include generated code here.
    // Checks ring of pixels, and populates "pattern".
    #include "fast2_gen.cu"

    if (mask_test(pattern)) {
        // Atomically append to corner buffer.
        int const icorn = atomicAdd(o_ncorners, 1);
        if (icorn < FAST_COUNT)
            o_corners[icorn] = xy;
    }
}

static void cufast(uchar1 const * data, int nx, int ny) {
    // Configure texture object.
    testImage.addressMode[0] = hipAddressModeClamp;
    testImage.addressMode[1] = hipAddressModeClamp;
    testImage.filterMode     = hipFilterModePoint;
    testImage.normalized     = false;

    // Create channel descriptor.
    hipChannelFormatDesc const format = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);

    // Allocate texture array.
    hipArray * buffer = NULL;
    hipMallocArray(&buffer, &format, nx, ny);

    // Populate texture array.
    hipMemcpyToArray(buffer, 0, 0, data, nx * ny * sizeof(uchar1), hipMemcpyHostToDevice);
    hipBindTextureToArray(testImage, buffer, format);

    // Allocate corner array 1.
    int2 * corners1 = NULL;
    hipMalloc(&corners1, sizeof(int2) * FAST_COUNT);

    // Allocate corner array 2.
    int2 * corners2 = NULL;
    hipMalloc(&corners2, sizeof(int2) * FAST_COUNT);

    // Allocate corner cursor 1.
    int * icorner1 = NULL;
    hipMalloc(&icorner1, sizeof(int));

    // Allocate corner cursor 2.
    int * icorner2 = NULL;
    hipMalloc(&icorner2, sizeof(int));

    // Reset corner cursors.
    int const zero = 0;
    hipMemcpy(icorner1, &zero, sizeof(zero), hipMemcpyHostToDevice);
    hipMemcpy(icorner2, &zero, sizeof(zero), hipMemcpyHostToDevice);

    // Create work grid 1.
    dim3 const dimBlock1(16, 16, 1);
    dim3 const dimGrid1((nx - (X_OFF * 2)) / dimBlock1.x, (ny - (Y_OFF * 2)) / dimBlock1.y, 1);

    // Warmup.
    fast1_kernel<<<dimGrid1, dimBlock1, 0>>>(corners1, icorner1);
    fast2_kernel<<<FAST_COUNT / NTHREADS, NTHREADS>>>(corners1, icorner1, corners2, icorner2);

    // Reset corner cursors.
    hipMemcpy(icorner1, &zero, sizeof(zero), hipMemcpyHostToDevice);
    hipMemcpy(icorner2, &zero, sizeof(zero), hipMemcpyHostToDevice);

    // Read number of corners.
    int ncorners1 = 0;
    int ncorners2 = 0;

    // Run kernels for time.

    long const time1 = time(NULL);

    for (int i = 0; i < REPEAT; i++) {
        hipMemcpy(icorner1, &zero, sizeof(zero), hipMemcpyHostToDevice);
        hipDeviceSynchronize();
        fast1_kernel<<<dimGrid1, dimBlock1, 0>>>(corners1, icorner1);
        hipDeviceSynchronize();
    }

    long const time2 = time(NULL);

    for (int i = 0; i < REPEAT; i++) {
        hipMemcpy(icorner2, &zero, sizeof(zero), hipMemcpyHostToDevice);
        hipDeviceSynchronize();
        fast2_kernel<<<FAST_COUNT / NTHREADS, NTHREADS>>>(corners1, icorner1, corners2, icorner2);
        hipDeviceSynchronize();
    }

    long const time3 = time(NULL);

    hipMemcpy(&ncorners1, icorner1, sizeof(ncorners1), hipMemcpyDeviceToHost);
    hipMemcpy(&ncorners2, icorner2, sizeof(ncorners2), hipMemcpyDeviceToHost);

    // Cleanup.
    hipFree(icorner2);
    hipFree(icorner1);
    hipFree(corners2);
    hipFree(corners1);
    hipFreeArray(buffer);

    // Calculate microseconds per kernel.
    int const us1 = (((time2 - time1) * 1000000) / REPEAT);
    int const us2 = (((time3 - time2) * 1000000) / REPEAT);

    // Report timing and number of corners.
    std::cerr << std::setw(8) << ncorners1 << " corners 1" << std::endl;
    std::cerr << std::setw(8) << ncorners2 << " corners 2" << std::endl;
    std::cerr << std::setw(8) << us1 << " microseconds 1" << std::endl;
    std::cerr << std::setw(8) << us2 << " microseconds 2" << std::endl;
}

int main(int argc, char **argv) {
    CVD::Image<CVD::byte> const fullImage = CVD::img_load("../images/shuttle.jpg");
    CVD::ImageRef const fullSize = fullImage.size();

    // Image size to keep for computation.
    int const nx = 2048;
    int const ny = 2048;
    CVD::ImageRef const keepSize(nx, ny);

    // Crop to sub-image.
    CVD::Image<CVD::byte> keepImage(keepSize);
    keepImage.copy_from(fullImage.sub_image(CVD::ImageRef(0, 0), keepSize));

    // Re-interpret image pointer.
    uchar1 const * const data = reinterpret_cast<uchar1 const *>(keepImage.data());

    // Test and benchmark CUFAST.
    cufast(data, nx, ny);

    return 0;
}
