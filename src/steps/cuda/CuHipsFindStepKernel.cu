// Copyright (C) 2011  Dmitri Nikulin, Monash University
//
// Permission is hereby granted, free of charge, to any person
// obtaining a copy of this software and associated documentation
// files (the "Software"), to deal in the Software without
// restriction, including without limitation the rights to use,
// copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the
// Software is furnished to do so, subject to the following
// conditions:
//
// The above copyright notice and this permission notice shall be
// included in all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
// EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES
// OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
// NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT
// HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY,
// WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR
// OTHER DEALINGS IN THE SOFTWARE.

#include <hip/hip_runtime.h>

#include <iostream>

namespace CVD {
namespace CL  {

#define THRESHOLD 3

// Parallel bit counting magic adapted from
// http://graphics.stanford.edu/~seander/bithacks.html#CountBitsSetParallel
__device__ uint bitcount32(uint v) {
    v = (v - ((v >> 1) & 0x55555555));
    v = ((v & 0x33333333) + ((v >> 2) & 0x33333333));
    v = ((((v + (v >> 4)) & 0xF0F0F0F) * 0x1010101) >> 24);
    return v;
}

__device__ uint bitcount64(ulong v) {
    uint const lo = (uint)(v);
    uint const hi = (uint)(v >> 32UL);
    return bitcount32(lo) + bitcount32(hi);
}

__device__ uint bitcount64_4(ulong4 t, ulong4 r) {
    return bitcount64(t.x & ~r.x) + bitcount64(t.y & ~r.y) + bitcount64(t.z & ~r.z) + bitcount64(t.w & ~r.w);
}

__global__ void hips_find_kernel(
    // N.B.: These uint8 are actually ulong4.
    ulong4 const * hashes1,  // T
    ulong4 const * hashes2,  // R
    int2         * matches,  // Pairs of indices into hashes1 and hashes2.
    uint         * imatch,   // Output number of hash1 matches.
    uint           nmatch    // Maximum number of matches.
) {

    // Use global work items for hash1, hash2 indices.
    uint   const ihash1  = (blockIdx.x * blockDim.x + threadIdx.x);
    uint   const ihash2  = (blockIdx.y * blockDim.y + threadIdx.y);

    // Read hashes.
    ulong4 const hash1   = hashes1[ihash1];
    ulong4 const hash2   = hashes2[ihash2];

    // Calculate error.
    uint   const error   = bitcount64_4(hash1, hash2);

    // Record match if within error threshold.
    if (error <= THRESHOLD) {
        uint const i = atomicAdd(imatch, 1);
        if (i < nmatch) {
            matches[i] = make_int2(ihash1, ihash2);
        }
    }
}

void hips_find(
    ulong4 const * hashes1,  // T
    ulong4 const * hashes2,  // R
    int2         * matches,  // Pairs of indices into hashes1 and hashes2.
    uint         * imatch,   // Output number of hash1 matches.
    uint           nmatch,   // Maximum number of matches.
    uint           np1,
    uint           np2
) {

    // Divide number of descriptors.
    unsigned int const np1_16 = (np1 / 16);
    unsigned int const np2_16 = (np2 / 16);

    // Prepare work sizes.
    dim3 blockDim(16, 16, 1);
    dim3 gridDim(np1_16, np2_16, 1);

    // Run kernel.
    std::cerr << "running kernel" << std::endl;
    hips_find_kernel<<<gridDim, blockDim>>>(hashes1, hashes2, matches, imatch, nmatch);
    std::cerr << "ran kernel" << std::endl;
}

} // namespace CL
} // namespace CVD
